#include "hip/hip_runtime.h"
#include "board.h"

using namespace cgol;

__global__ void step(int *arr, int *result, size_t N, int width) {
    int index =  blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

   for(int i = index; i < N; i += stride)
   {
       
   }
}


hipDeviceProp_t getDetails(int deviceId)
{
    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, deviceId);
    return props;
}


#define multi 20
void launcher(board *mb) {

    int deviceId;
    hipGetDevice(&deviceId);
    hipDeviceProp_t props = getDetails(deviceId);

    size_t N = (mb->width * mb->height);
    size_t size = sizeof(int) * N;

    int *result;
    int *d_result;
    int *d_input;

    result = (int*)malloc(size);
    
    // allocate device memory for the maps
    hipMalloc((void **)&d_input, size);
    hipMalloc((void **)&d_result, size);

    hipMemcpy(d_input, mb->arr, size, hipMemcpyHostToDevice);


    // calculate kernel configuaration
    int threads_per_block = 512;
    printf("number of sms :%d \n", props.multiProcessorCount);
    int number_of_blocks = props.multiProcessorCount * multi;

    //create error variables
    hipError_t step_error;
    hipError_t asyncErr;

    // call the kernel
    step<<<threads_per_block, number_of_blocks>>>(d_input, d_result, N, mb->width);

    // copy the data back
    hipMemcpy(result, d_result, size, hipMemcpyDeviceToHost);

    // check for errors
    step_error = hipGetLastError();
    if(step_error != hipSuccess) printf("Error: %s\n", hipGetErrorString(step_error));

    asyncErr = hipDeviceSynchronize();
    if(asyncErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(asyncErr));

    // clean up
    hipFree(d_result); hipFree(d_input);
    free(result);
}