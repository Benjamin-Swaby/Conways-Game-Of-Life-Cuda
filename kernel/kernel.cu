#include "hip/hip_runtime.h"
#include "board.h"

#include <iostream>

using namespace cgol;

__global__ void step(int *arr, int *result, size_t N, int width) {
    int index =  blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

   for(int i = index; i < N; i += stride)
   {
       //TODO check all cells around the target cell 
       // arr[i] is target cell       
        result[i] = arr[i];
        int live_neighbours = 0;
        int target;

        int neighbour_indexes[8];

        neighbour_indexes[0] = (i - width) - 1; // top left
        neighbour_indexes[1] = (i - width); // top
        neighbour_indexes[2] = (i - width) + 1; // top right

        neighbour_indexes[3] = (i + 1); // right
        neighbour_indexes[4] = (i - 1); // left

        neighbour_indexes[5] = (i + width) - 1; // bottom left
        neighbour_indexes[6] = (i + width); // bottom
        neighbour_indexes[7] = (i + width) + 1; // bottom right


        // if the top left isn't at the end of the line or before the array
        if (!(neighbour_indexes[0] < 0 || neighbour_indexes[0] % (width - 1) == 0)) {
            if (arr[neighbour_indexes[0]]) {
                live_neighbours++;
            }
        }

        // if the top one does exist 
        if (!(neighbour_indexes[1] < 0)) {
            if (arr[neighbour_indexes[1]]) {
                live_neighbours++;
            }
        }

        if(!(neighbour_indexes[2] < 0 || neighbour_indexes[2] % width == 0)) {
            if (arr[neighbour_indexes[2]]) {
                live_neighbours++;
            }
        }

        // -----------------------------------------


        if (live_neighbours < 2) { // dies of underpopulation
            result[i] = 0;
        }else if (live_neighbours == 2 || live_neighbours == 3) // lives on to next gen
        {
            result[i] = 1;
        }else if (live_neighbours > 3) // dies of overpopulation
        {
            result[i] = 0;
        }else if (live_neighbours == 3 && !arr[i]) // a cell is birthed by 3 neighbours
        {
            result[i] = 1;
        }else {  // cell dies 
            result[i] = 0; 
        }
        
        
        

   }
}


hipDeviceProp_t getDetails(int deviceId)
{
    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, deviceId);
    return props;
}


#define multi 20
void launcher(board *mb) {

    int deviceId;
    hipGetDevice(&deviceId);
    hipDeviceProp_t props = getDetails(deviceId);

    size_t N = (mb->width * mb->height);
    size_t size = sizeof(int) * N;

    int *result;
    int *d_result;
    int *d_input;

    result = (int*)malloc(size);
    
    // allocate device memory for the maps
    hipMalloc((void **)&d_input, size);
    hipMalloc((void **)&d_result, size);

    hipMemcpy(d_input, mb->arr, size, hipMemcpyHostToDevice);


    // calculate kernel configuaration
    int threads_per_block = 512;
    printf("number of sms :%d \n", props.multiProcessorCount);
    int number_of_blocks = props.multiProcessorCount * multi;

    //create error variables
    hipError_t step_error;
    hipError_t asyncErr;

    // call the kernel
    step<<<threads_per_block, number_of_blocks>>>(d_input, d_result, N, mb->width);

    // copy the data back
    hipMemcpy(result, d_result, size, hipMemcpyDeviceToHost);

    // check for errors
    step_error = hipGetLastError();
    if(step_error != hipSuccess) printf("Error: %s\n", hipGetErrorString(step_error));

    asyncErr = hipDeviceSynchronize();
    if(asyncErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(asyncErr));

    // create a new board and link it to the head / previous board
    board *result_board = new board(mb->width, mb->height);
    result_board->arr = result;
    result_board->position = mb->position + 1;
    mb->next = result_board;

    // clean up
    hipFree(d_result); hipFree(d_input);
}