#include "hip/hip_runtime.h"
#include "board.h"

#include <iostream>

using namespace cgol;

__global__ void step(int *arr, int *result, size_t N, int width) {
    int index =  blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

   for(int i = index; i < N; i += stride)
   {
        int live_neighbours = 0;
        int neighbour_indexes[8];

        neighbour_indexes[0] = (i - width) - 1; // top left
        neighbour_indexes[1] = (i - width); // top
        neighbour_indexes[2] = (i - width) + 1; // top right

        neighbour_indexes[3] = (i - 1); // left
        neighbour_indexes[4] = (i + 1); // right

        neighbour_indexes[5] = (i + width) - 1; // bottom left
        neighbour_indexes[6] = (i + width); // bottom
        neighbour_indexes[7] = (i + width) + 1; // bottom right


        // if the top left isn't at the end of the line or before the array
        if (!(neighbour_indexes[0] < 0 || neighbour_indexes[0] % width == (width - 1))) {
            if (arr[neighbour_indexes[0]]) {
                live_neighbours++;
            }
        }

        // if the top one does exist 
        if (!(neighbour_indexes[1] < 0)) {
            if (arr[neighbour_indexes[1]]) {
                live_neighbours++;
            }
        }

        // if the top right isn't at the start of a line or before the array
        if (!(neighbour_indexes[2] < 0 || neighbour_indexes[2] % width == 0)) {
            if (arr[neighbour_indexes[2]]) {
                live_neighbours++;
            }
        }

        // if the left isn't at the end of a line
        if (!(neighbour_indexes[3] % width == (width - 1)) || neighbour_indexes[3] < 0) {
            if (arr[neighbour_indexes[3]]) {
                live_neighbours++;
            }
        }

        // if the right isn't at the start of the next line
        if (!(neighbour_indexes[4] % width == 0) || neighbour_indexes[4] > N) {
            if (arr[neighbour_indexes[4]]) {
                live_neighbours++;
            }
        }

        // if the bottom left isn't at the end of a line
        if (!(neighbour_indexes[5] > N || neighbour_indexes[5] % width == (width - 1))) {
            if (arr[neighbour_indexes[5]]) {
                live_neighbours++;
            }
        }

        // if the bottom one isn't out of the array
        if (neighbour_indexes[6] < N) {
            if (arr[neighbour_indexes[6]]) {
                live_neighbours++;
            }
        }

        // if the bottom right isn't at the start of a line or out of the array
        if (!(neighbour_indexes[7] > N || neighbour_indexes[7] % width == 0)) {
            if (arr[neighbour_indexes[7]]) {
                live_neighbours++;
            }
        }

        // -----------------------------------------
        

        //printf("Cell %d has %d ln \n" , i , live_neighbours);

        if (arr[i] && (live_neighbours == 2 || live_neighbours == 3)) {
            result[i] = 1;
        }else if (!arr[i] && (live_neighbours == 3))
        {
            result[i] = 1;
        }else {
            if(arr[i]) {
                result[i] = 0;
            }else if (!arr[i]) {
                result[i] = 0;
            }
        }
        
        
        
        

   }
}


hipDeviceProp_t getDetails(int deviceId)
{
    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, deviceId);
    return props;
}


#define multi 20
void launcher(board *mb) {

    int deviceId;
    hipGetDevice(&deviceId);
    hipDeviceProp_t props = getDetails(deviceId);

    size_t N = (mb->width * mb->height);
    size_t size = sizeof(int) * N;

    int *result;
    int *d_result;
    int *d_input;

    result = (int*)malloc(size);
    
    // allocate device memory for the maps
    hipMalloc((void **)&d_input, size);
    hipMalloc((void **)&d_result, size);

    hipMemcpy(d_input, mb->arr, size, hipMemcpyHostToDevice);


    // calculate kernel configuaration
    int threads_per_block = 512;
    //printf("number of sms :%d \n", props.multiProcessorCount);
    int number_of_blocks = props.multiProcessorCount * multi;

    //create error variables
    hipError_t step_error;
    hipError_t asyncErr;

    // call the kernel
    step<<<threads_per_block, number_of_blocks>>>(d_input, d_result, N, mb->width);

    // copy the data back
    hipMemcpy(result, d_result, size, hipMemcpyDeviceToHost);

    // check for errors
    step_error = hipGetLastError();
    if(step_error != hipSuccess) printf("Error: %s\n", hipGetErrorString(step_error));

    asyncErr = hipDeviceSynchronize();
    if(asyncErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(asyncErr));

    // create a new board and link it to the head / previous board
    board *result_board = new board(mb->width, mb->height);
    result_board->arr = result;
    result_board->position = mb->position + 1;
    
    // link backwards and forwards
    result_board->prev = mb;
    mb->next = result_board;

    // clean up
    hipFree(d_result); hipFree(d_input);
}